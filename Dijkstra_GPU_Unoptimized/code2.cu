
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>


#define TIMER_CREATE(t)             \
        hipEvent_t t##_start, t##_end;     \
        hipEventCreate(&t##_start);        \
        hipEventCreate(&t##_end);


#define TIMER_START(t)                \
        hipEventRecord(t##_start);         \
        hipEventSynchronize(t##_start);    \


#define TIMER_END(t)                             \
        hipEventRecord(t##_end);                      \
        hipEventSynchronize(t##_end);                 \
        hipEventElapsedTime(&t, t##_start, t##_end);  \
        hipEventDestroy(t##_start);                   \
        hipEventDestroy(t##_end);


//Function to check for errors
inline hipError_t checkCuda(hipError_t result) 
{
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != cudaSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
			exit(-1);
		}
	#endif
	return result;
}

//Number of vertices
#define vertices 5000 

//Number of edges per vertex
#define Edge_per_node 4999

//Used to define weights on each edge.
#define Maximum_weight 5

//Value for infinity
#define infinity 10000000

//Kernel call to inititialize all node weights to infinity except for the source node. We mark the source node as settled after this point
__global__ void Initializing(int *node_weight_array, int *mask_array, int Source) // CUDA kernel
{
	int id = blockIdx.x*blockDim.x+threadIdx.x; // Get global thread ID
	if(id<vertices)
	{
		if(id==Source)
		{
			node_weight_array[id]=0;
			mask_array[id]=1;
		}
		else
		{
			node_weight_array[id]=infinity;
			mask_array[id]=0;
		}
	}
}


//Kernel Call to choose a a node which is relaxed and settled and to relax the outgoing edges of each settled node.
__global__ void Minimum(int *mask_array,int *vertex_array,int *vertex_array_copy, int *node_weight_array, int *edge_array, int *edge_weight_array, int *min)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x; // Get global thread ID
	//Iterative variables
	int i,n,t;	
	if(id<vertices)
	{
		if(mask_array[id]==1)
		{
			t=vertex_array_copy[id];	
			for(i=t*Edge_per_node;i<t*Edge_per_node+Edge_per_node;i++)
			{
				n=edge_array[i];
				if(mask_array[n]!=1)
				{
					atomicMin(&node_weight_array[n],node_weight_array[id]+edge_weight_array[i]);
					atomicMin(&min[0],node_weight_array[n]);
					vertex_array_copy[id]=n;
					break;
				}
			}
		}
	}		
}

//Kernel call to mark all the settled nodes
__global__ void Relax(int *mask_array,int *node_weight_array,int *min)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x; // Get global thread ID
	if(id<vertices)
	{
		if(mask_array[id]!=1 && node_weight_array[id]==min[0])
		{
			mask_array[id]=1;
		}
	}	
}

int main( int argc, char* argv[] )
{

	//Size of the Vertex array
	size_t vertex_array_size = vertices*sizeof(int);
	
	//Size of the edge array and edge_weight array
	size_t edge_array_size = vertices*Edge_per_node*sizeof(int);

	//Intializing the vertex array
	int *vertex_array = (int*)malloc(vertex_array_size); 

	//Intializing the vertex array
	int *vertex_array_copy = (int*)malloc(vertex_array_size); 

	//Initializing a copy of the vertex array
	int *vertex_copy = (int*)malloc(vertex_array_size); 

	//Intializing the edge array
	int *edge_array=(int*)malloc(edge_array_size);

	//Initializing edge_weight_array which stores the weights of each edge
	int *edge_weight_array = (int*)malloc(edge_array_size);

	//Initializing Node weight array which stores the value for the current weight to reach the node
	int *node_weight_array = (int*)malloc(vertex_array_size);

	//Array to mark if a node is settled or not
	int *mask_array = (int*)malloc(vertex_array_size);

	//Iterative operator
	int i,j,k;  

	printf("Populating Vertex Array....\n");

	//Setting node number in vertex_array
	for(i=0;i<vertices;i++)
	{
		vertex_array[i]=i;
	}
	
	//Setting the seed of the RNG to system clock
	srand(time(NULL));

	//temp variable
	int temp;

	//Adding random edges to each node while avoiding self edge
	memcpy(vertex_copy,vertex_array,vertex_array_size);
	memcpy(vertex_array_copy,vertex_array,vertex_array_size);

	printf("Populating Edge Array....\n");

	//We give each node random edges and store them in the increasing order of weights in the edge array.
	for(i=0;i<vertices;i++)
	{
		//Function to jumble the nodes in the vertex array and assign them to each node
		for(j=vertices-1;j>0;j--)
		{		
			k=rand()%(j+1);
			temp = vertex_copy[j];
			vertex_copy[j]=vertex_copy[k];
			vertex_copy[k]=temp;
		}

		for(j=0;j<Edge_per_node;j++)
		{
			if(vertex_copy[j]==i)
			{
				j=j+1;
				edge_array[i*Edge_per_node+(j-1)]= vertex_copy[j];			
			}
			else
			{
				edge_array[i*Edge_per_node+j]= vertex_copy[j];			
			}
		}

	}

/*	
	//Can be uncommented to see the edges of each node
	printf("=== Initial edges===\n");
	for(i=0;i<vertices*Edge_per_node;i++)
	{
		printf("E[%d]= %d\n",i,edge_array[i]);
	}
*/	

	printf("Adding Weights to each edge...\n");

	//Adding weights to the edge_weight array
	for(i=0;i<vertices;i++)
	{
		int a = rand()%Maximum_weight+1;
		int b = rand()%Maximum_weight+1;
		for(j=0;j<Edge_per_node;j++)
		{
			edge_weight_array[i*Edge_per_node+j]=a+j*b;
		}
	}

/*	
	//Can be uncommented to see the edge weight of each edge
	printf("=== Initial edge weight weight===\n");
	for(i=0;i<vertices*Edge_per_node;i++)
	{
		printf("W[%d]= %d\n",i,edge_weight_array[i]);
	}
*/

	//Initializing gpu variables
	int *gpu_vertex_array;
	int *gpu_vertex_array_copy;
	int *gpu_edge_array;
	int *gpu_edge_weight_array;
	int *gpu_node_weight_array;
	int *gpu_mask_array;

	//Allocating memory to the gpu variables
	checkCuda(hipMalloc((void**)&gpu_vertex_array,vertex_array_size));
	checkCuda(hipMalloc((void**)&gpu_vertex_array_copy,vertex_array_size));
	checkCuda(hipMalloc((void**)&gpu_node_weight_array,vertex_array_size));
	checkCuda(hipMalloc((void**)&gpu_mask_array,vertex_array_size));
	checkCuda(hipMalloc((void**)&gpu_edge_array,edge_array_size));
	checkCuda(hipMalloc((void**)&gpu_edge_weight_array,edge_array_size));

	//Copying memory from Host to Device	
	checkCuda(hipMemcpy(gpu_vertex_array,vertex_array,vertex_array_size,hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(gpu_vertex_array_copy,vertex_array_copy,vertex_array_size,hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(gpu_node_weight_array,node_weight_array,vertex_array_size,hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(gpu_mask_array,mask_array,vertex_array_size,hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(gpu_edge_array,edge_array,edge_array_size,hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(gpu_edge_weight_array,edge_weight_array,edge_array_size,hipMemcpyHostToDevice));

	//Setting the Block and Grid Size
	int blockSize, gridSize;
	blockSize=1024;
	gridSize = (int)ceil((float)vertices/blockSize); // Number of thread blocks in grid

	printf("Beginning Optimized Djikstra Algorithm\n");

	//Starting timer
	float start_time;
	TIMER_CREATE(start_time);
	TIMER_START(start_time);

	//Kernel call to initialize all the node weights. We provide the source node 0 
	Initializing<<<gridSize, blockSize>>>(gpu_node_weight_array,gpu_mask_array, 0);
	hipError_t err = hipGetLastError();
	/*	
	if (err != cudaSuccess) checkCuda(cudaMemcpy(node_weight_array,gpu_node_weight_array,vertex_array_size,cudaMemcpyDeviceToHost));
	{
		printf("Error: %s\n", cudaGetErrorString(err));
	}
	*/

/*
	//Can be uncommented to see the initial weights of each node
	checkCuda(cudaMemcpy(node_weight_array,gpu_node_weight_array,vertex_array_size,cudaMemcpyDeviceToHost));	
	printf("=== Initial node weight===\n");
	for(i=0;i<vertices;i++)
	{
		printf("NW[%d]= %d\n ",i,node_weight_array[i]);
	}
*/

	//Variable min used to store the minimum node wieght of the relaxed nodes and use this node to relax all of its edges
	int *min=(int*)malloc(2*sizeof(int));
	min[0]=0;
	min[1]=0;

	//GPU variable to store min value
	int *gpu_min;
	checkCuda(hipMalloc((void**)&gpu_min,2*sizeof(int)));

	//Begin the relax calls of the algorithm
	while(min[0]<infinity)
	{
		min[0] = infinity;
		checkCuda(hipMemcpy(gpu_min,min,sizeof(int),hipMemcpyHostToDevice));

		Minimum<<<gridSize, blockSize>>>(gpu_mask_array,gpu_vertex_array,gpu_vertex_array_copy,gpu_node_weight_array,gpu_edge_array,gpu_edge_weight_array,gpu_min);
		/*
		if (err != cudaSuccess) checkCuda(cudaMemcpy(node_weight_array,gpu_node_weight_array,vertex_array_size,cudaMemcpyDeviceToHost));
		{
			printf("Error: %s\n", cudaGetErrorString(err));
		}
		*/
		Relax<<<gridSize, blockSize>>>(gpu_mask_array,gpu_node_weight_array,gpu_min);
		/*	
		if (err != cudaSuccess) checkCuda(cudaMemcpy(node_weight_array,gpu_node_weight_array,vertex_array_size,cudaMemcpyDeviceToHost));
		{
			printf("Error: %s\n", cudaGetErrorString(err));
		}
		*/

/*
		//Can be uncommented to see the node weight and Dijkistra's Algorithm being performed ste by step
		checkCuda(cudaMemcpy(node_weight_array,gpu_node_weight_array,vertex_array_size,cudaMemcpyDeviceToHost));
		for(i=0;i<vertices;i++)
		{
			printf("NW[%d]= %d\n ",i,node_weight_array[i]);
		}
*/

		checkCuda(hipMemcpy(min,gpu_min,2*sizeof(int),hipMemcpyDeviceToHost));	
	}

	//End timer
	TIMER_END(start_time);
	printf("Kernel Execution Time: %f ms\n",start_time);


	//Coppying the the final node weights from the Device to Host
	checkCuda(hipMemcpy(node_weight_array,gpu_node_weight_array,vertex_array_size,hipMemcpyDeviceToHost));


/*
	//Can be uncommented to see the final shortes distance of all node from Source Node
	printf("=== Final node weight===\n");
	for(i=0;i<vertices;i++)
	{
		printf("NW[%d]= %d\n ",i,node_weight_array[i]);
	}
*/

	hipFree(gpu_vertex_array);
	hipFree(gpu_node_weight_array);
	hipFree(gpu_edge_array);
	hipFree(gpu_edge_weight_array);
	hipFree(gpu_mask_array);

	free(vertex_array);
	free(node_weight_array);
	free(edge_array);
	free(edge_weight_array);
	free(mask_array);

	return 0;
}
