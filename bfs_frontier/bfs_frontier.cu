
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define NUM_NODES 1024

// Declaration of a structure 
typedef struct {
	int startIndex; // starting index in Adj list	
	int numberOfNeighbors; // number of neighbors of each vertices
} Node;

__global__ void bfs_optimized(Node *gpu_vertex, int *gpu_neighbors, bool *gpu_frontier, bool *gpu_visited, int *gpu_cost, bool *gpu_done) {

	// ThreadID
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	
	// boundary condition for threadID
	if (threadId > NUM_NODES)
		*gpu_done = false;
	
	// checking condition for frontier and visited node array
	if (gpu_frontier[threadId] == true && gpu_visited[threadId] == false) { 
		
		// Init	
		gpu_frontier[threadId] = false;
		gpu_visited[threadId] = true;
	
		// assign values from array
		int startPoint = gpu_vertex[threadId].startIndex;
		int endPoint = startPoint + gpu_vertex[threadId].numberOfNeighbors;

		// traverse to the neighbors for every vertex
		for (int i = startPoint; i < endPoint; i++) {
			int neighbor = gpu_neighbors[i];

			// check visited mark and increase cost
			if (gpu_visited[neighbor] == false) {
				gpu_cost[neighbor] = gpu_cost[threadId] + 1;
				gpu_frontier[neighbor] = true;
				*gpu_done = false;

			}

		}

	}

}

// Main method
int main(int argc, char* argv[]) {

	// Kernel launch parameters
        int numberOfThreads = 1024;
	int numberOfBlocks = NUM_NODES/numberOfThreads;

	// Intialization of struct and neighbors array
	Node vertex[NUM_NODES];
	int edges[NUM_NODES];

	// populate the graph
        for(int i=0;i<NUM_NODES;i++) {
                vertex[i].numberOfNeighbors = 1;//(rand() % 5)+1;
        }

        vertex[0].startIndex = 0;
        for(int j=1;j<NUM_NODES;j++) {
                vertex[j].startIndex = vertex[j-1].startIndex + vertex[j-1].numberOfNeighbors;
        }

 	for(int k=0;k<NUM_NODES;k++) {
                edges[k] = k+1;
        }

	hipSetDevice(0);

	// Time variable
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Intitalization of array for frontier and visited nodes and costpath
	bool frontierArray[NUM_NODES] = { false };
	bool visitedNodes[NUM_NODES] = { false };
	int costOfPath[NUM_NODES] = { 0 };

	int source = 0;
	frontierArray[source] = true;

	// GPU variable declaration
	Node* gpu_vertex;
	int* gpu_neighbors;
	bool* gpu_frontier;
	bool* gpu_visited;
	int* gpu_cost;
	bool* gpu_done;

	// GPU memory allocation
	hipMalloc((void**)&gpu_vertex, sizeof(Node)*NUM_NODES);
	hipMalloc((void**)&gpu_neighbors, sizeof(Node)*NUM_NODES);
	hipMalloc((void**)&gpu_frontier, sizeof(bool)*NUM_NODES);
	hipMalloc((void**)&gpu_visited, sizeof(bool)*NUM_NODES);
	hipMalloc((void**)&gpu_cost, sizeof(int)*NUM_NODES);
	hipMalloc((void**)&gpu_done, sizeof(bool));

	// Transfer of data from CPU to GPU
	hipMemcpy(gpu_vertex, vertex, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_neighbors, edges, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_frontier, frontierArray, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_visited, visitedNodes, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_cost, costOfPath, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	bool cpu_done;

	hipEventRecord(start, 0);
	int Kernel_call_count = 0;

	do {
		Kernel_call_count++;
		cpu_done = true;
		hipMemcpy(gpu_done, &cpu_done, sizeof(bool), hipMemcpyHostToDevice);
	
		// Kernel call
		bfs_optimized<<<numberOfBlocks, numberOfThreads>>>(gpu_vertex, gpu_neighbors, gpu_frontier, gpu_visited, gpu_cost, gpu_done);

		hipMemcpy(&cpu_done, gpu_done , sizeof(bool), hipMemcpyDeviceToHost);

	} while (!cpu_done);

	// Copy final results from GPU to CPU
	hipMemcpy(costOfPath, gpu_cost, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);
	
	printf("Kernel call count: %d\n", Kernel_call_count);

	hipEventRecord(stop, 0);

	hipEventElapsedTime(&time, start, stop);
	printf("Parallel Job execution time: %.2f ms\n", time);

	hipFree(gpu_vertex);
	hipFree(gpu_neighbors);
	hipFree(gpu_frontier);
	hipFree(gpu_visited);
	hipFree(gpu_cost);
	hipFree(gpu_done);

	return 0;
}
